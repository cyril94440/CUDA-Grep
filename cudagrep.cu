#include "hip/hip_runtime.h"
#include <dirent.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

using namespace std;

void sendGPU(char *A, unsigned int indexOfLine[], unsigned int &iLine); //send to GPU routine

char *d_word; //GPU MEMORY word pattern pointer.
char word[100];


// EDIT THIS ACCORDING TO YOUR GPU CAPABILITIES
const int MAX_THREADS = 1024;
const int MAX_BLOCKS = 1024;

// GREP GPU KERNEL
__global__ void GrepKernel(char *A, bool *R, char *wordD, unsigned int *indexOfLine)
{
    int row = threadIdx.x + blockIdx.x * gridDim.x;
    
    if(indexOfLine[row+2]!=0) //To be sure that there is something to check.
    {
        int indexWord = 0;
        bool matching = false;
        bool matched = false;
        
        bool starting = false; //^ REGEX
        if(wordD[0]=='^')
        {
            indexWord=1;
            matching = true;
            starting = true;
        }
        
        for(int j=0;j<(indexOfLine[row+1]-indexOfLine[row]);j++) //Check each characters
        {
            if(A[indexOfLine[row]+j]=='\0')//End of the line reached
                break;
            
            if(((A[indexOfLine[row]+j]==wordD[indexWord])&&(matching==true || indexWord==0))||(wordD[indexWord]=='.'))//Letter match
            {
                matching=true;
                indexWord++;
                if(wordD[indexWord]=='\0')
                {
                    matched = true;
                    break;
                }
            }
            else if(matching==true)//Was matching and letter does not seem to match
            {
                if(wordD[indexWord]=='$') //$ REGEX
                {
                    if(j+1==(indexOfLine[row+1]-indexOfLine[row]))
                    {
                        matched=true;
                        break;
                    }
                    else //It does not match anymore RESTART matching
                    {
                        indexWord=0;
                        matching=false;
                    }
                }
                else if(wordD[indexWord]=='*' && A[indexOfLine[row]+j]!=' ')//* REGEX
                {
                    if(wordD[indexWord+1]==A[indexOfLine[row]+j+1])
                    {
                        indexWord++;
                    }
                }
                else if(starting) //^REGEX
                    break;
                else //It does not match anymore RESTART matching
                {
                    indexWord=0;
                    matching=false;
                }
            }
        }
        
        if(matched)
            R[row]=true;
        else
            R[row]=false;
    }
    
}

int main(int argc, const char * argv[])
{
    strcpy(word, argv[2]);
    
    //LOAD word INTO DEVICE MEMORY
    hipMalloc((void**)&d_word, 100);
    hipMemcpy(d_word, word, 100, hipMemcpyHostToDevice);
    
    //ALLOC ARRAY
    char *A=(char*)malloc(2000000000);
    A[0]='\0';
    
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    
    //OPEN FILE
    fp = fopen(argv[1], "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    
    unsigned int indexOfLine[(MAX_THREADS*MAX_BLOCKS)+2];
    indexOfLine[0]=0;
    
    unsigned int iLine = 0;
    
    while ((read = getline(&line, &len, fp)) != -1)//Line per line read
    {
        int index = indexOfLine[iLine];
        int i=0;
        for(i;i>-1;i++)//Fill the array
        {
            if(line[i]=='\0')
                break;
            
            A[index+i]=line[i];
        }
        iLine++;
        
        indexOfLine[iLine]=i+index; //Store the index of the started line.
        
        if(iLine>=MAX_THREADS*MAX_BLOCKS)//MAX Amount of lines reached so send to the GPU
        {
            indexOfLine[iLine+1]=2;
            sendGPU(A,indexOfLine,iLine);
        }
    }
    
    
    if (line)
        free(line);
    
    //File fully read, last send to the GPU
    for(int i=iLine+1;i<(MAX_BLOCKS*MAX_THREADS)+2;i++)
        indexOfLine[i]=0;
    
    sendGPU(A, indexOfLine,iLine);
    
    //FREE the memory
    hipFree(d_word);
    free(A);
    
    return 0;
}

void sendGPU(char *A, unsigned int indexOfLine[], unsigned int &iLine)
{
    //Allocations
    unsigned int size = indexOfLine[iLine];
    bool *R=(bool*)malloc(iLine*sizeof(bool));
    
    //SEND TO GPU ROUTINE
    
    //LOAD A INTO DEVICE MEMORY
    char *d_A;
    hipError_t err = hipMalloc((void**)&d_A, size);
    //printf("CUDA malloc A: %s\n",hipGetErrorString(err));
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    //printf("Copy A to device: %s\n",hipGetErrorString(err));
    
    //LOAD indexOfLine INTO DEVICE MEMORY
    unsigned int *d_indexOfLine;
    err = hipMalloc((void**)&d_indexOfLine, ((MAX_THREADS*MAX_BLOCKS)+2)*sizeof(unsigned int));
    err = hipMemcpy(d_indexOfLine, indexOfLine, ((MAX_THREADS*MAX_BLOCKS)+2)*sizeof(unsigned int), hipMemcpyHostToDevice);
    
    //CREATE R FOR RESULTS
    bool *d_R;
    err = hipMalloc((void**)&d_R, iLine*sizeof(bool));
    //printf("CUDA malloc C: %s\n",hipGetErrorString(err));
    
    // Invoke kernel
    dim3 dimBlock(MAX_THREADS,1);
    dim3 dimGrid(MAX_BLOCKS,1);
    GrepKernel<<<dimGrid, dimBlock>>>(d_A, d_R, d_word, d_indexOfLine);
    
    //Wait that the GPU work is over.
    err = hipDeviceSynchronize();
    
    
    //printf("Run kernel: %s\n", hipGetErrorString(err));
    
    // Read R from device memory
    err = hipMemcpy(R, d_R, iLine*sizeof(bool), hipMemcpyDeviceToHost);
    //printf("Copy R off of device: %s\n",hipGetErrorString(err));
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_R);
    hipFree(d_indexOfLine);
    
    // Display matched lines
    for(int i=0;i<iLine;i++)
    {
        if(R[i])
        {
            for(int j=0;j<(indexOfLine[i+1]-indexOfLine[i]);j++)
            {
                char letter = A[indexOfLine[i]+j];
                if(letter=='\0')
                    break;
                else
                    printf("%c",letter);
            }
            
        }
    }
    
    
    
    //Reset memory and counter.
    free(R);
    iLine=0;
    
}
